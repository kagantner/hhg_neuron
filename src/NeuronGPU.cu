#include "hip/hip_runtime.h"
#include "NeuronGPU.h"
#include <hip/hip_runtime.h>
#include <cmath>
#include <stdexcept>
#include <algorithm>
#include <vector>

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

// --- CUDA Error Handling Macro ---
#define CUDA_CHECK(err) { \
    hipError_t e = (err); \
    if (e != hipSuccess) { \
        throw std::runtime_error(std::string("CUDA Error: ") + hipGetErrorString(e)); \
    } \
}

// --- Device-side (GPU) HH Model Functions ---

// Note: These functions are marked with __device__ to indicate they run on the GPU.
// They are direct translations of the HHModel class methods.

__device__ double alpha_m_gpu(double V) {
    double V_shifted = V + 40.0;
    if (abs(V_shifted) < 1e-5) return 1.0;
    return 0.1 * V_shifted / (1.0 - exp(-V_shifted / 10.0));
}

__device__ double beta_m_gpu(double V) {
    return 4.0 * exp(-(V + 65.0) / 18.0);
}

__device__ double alpha_h_gpu(double V) {
    return 0.07 * exp(-(V + 65.0) / 20.0);
}

__device__ double beta_h_gpu(double V) {
    return 1.0 / (1.0 + exp(-(V + 35.0) / 10.0));
}

__device__ double alpha_n_gpu(double V) {
    double V_shifted = V + 55.0;
    if (abs(V_shifted) < 1e-5) return 0.1;
    return 0.01 * V_shifted / (1.0 - exp(-V_shifted / 10.0));
}

__device__ double beta_n_gpu(double V) {
    return 0.125 * exp(-(V + 65.0) / 80.0);
}

// --- Main CUDA Kernel ---

__global__ void update_kernel(
    double* V_m, double* m, double* h, double* n,
    const double* I_inj,
    int n_seg, double dt, double g_a, double surface_area) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_seg) return;

    // Biophysical constants (must be defined within device code)
    const double C_m_density = 1.0;
    const double g_Na_density = 120.0;
    const double g_K_density = 36.0;
    const double g_L_density = 0.3;
    const double E_Na = 50.0;
    const double E_K = -77.0;
    const double E_L = -54.387;

    // --- Step 1: Calculate axial current ---
    double v_prev = V_m[i]; // Store current voltage
    double v_left = (i > 0) ? V_m[i - 1] : v_prev;
    double v_right = (i < n_seg - 1) ? V_m[i + 1] : v_prev;
    double I_axial = g_a * (v_left - v_prev) + g_a * (v_right - v_prev);

    double total_current_uA = I_inj[i] + I_axial;
    double I_stim_density = (surface_area > 0) ? total_current_uA / surface_area : 0.0;

    // --- Step 2: Update membrane potential ---
    double I_Na = g_Na_density * m[i] * m[i] * m[i] * h[i] * (v_prev - E_Na);
    double I_K = g_K_density * n[i] * n[i] * n[i] * n[i] * (v_prev - E_K);
    double I_L = g_L_density * (v_prev - E_L);

    double dV = (I_stim_density - I_Na - I_K - I_L) / C_m_density;
    double new_V = v_prev + dV * dt;

    // --- Step 3: Update gating variables ---
    double am = alpha_m_gpu(new_V);
    double bm = beta_m_gpu(new_V);
    double ah = alpha_h_gpu(new_V);
    double bh = beta_h_gpu(new_V);
    double an = alpha_n_gpu(new_V);
    double bn = beta_n_gpu(new_V);

    m[i] = am / (am + bm) + (m[i] - am / (am + bm)) * exp(-dt * (am + bm));
    h[i] = ah / (ah + bh) + (h[i] - ah / (ah + bh)) * exp(-dt * (ah + bh));
    n[i] = an / (an + bn) + (n[i] - an / (an + bn)) * exp(-dt * (an + bn));

    // --- Step 4: Store new voltage ---
    V_m[i] = new_V;
}

// --- Host-side Class Implementation ---

// Definition of the struct holding GPU pointers
struct NeuronGPU_Data {
    double* d_V_m;
    double* d_m;
    double* d_h;
    double* d_n;
    double* d_I_inj;
    double g_a;
    double surface_area;
};

NeuronGPU::NeuronGPU(int num_segments, double length, double diameter, double Ra) {
    this->num_segments = num_segments;
    d_data = new NeuronGPU_Data();
    injected_currents_uA.resize(num_segments, 0.0);

    // Calculate geometry and conductance
    d_data->surface_area = M_PI * diameter * length;
    if (num_segments > 1) {
        double cross_area = M_PI * (diameter / 2.0) * (diameter / 2.0);
        double R_axial_kohm = (Ra * length / cross_area) / 1000.0;
        d_data->g_a = (R_axial_kohm > 0) ? 1.0 / R_axial_kohm : 0.0;
    } else {
        d_data->g_a = 0;
    }

    // Allocate GPU memory
    size_t size = num_segments * sizeof(double);
    CUDA_CHECK(hipMalloc(&d_data->d_V_m, size));
    CUDA_CHECK(hipMalloc(&d_data->d_m, size));
    CUDA_CHECK(hipMalloc(&d_data->d_h, size));
    CUDA_CHECK(hipMalloc(&d_data->d_n, size));
    CUDA_CHECK(hipMalloc(&d_data->d_I_inj, size));

    // Initialize state on host
    std::vector<double> h_V_m(num_segments, -65.0);
    std::vector<double> h_m(num_segments);
    std::vector<double> h_h(num_segments);
    std::vector<double> h_n(num_segments);

    double V_init = -65.0;
    double m_init = alpha_m_gpu(V_init) / (alpha_m_gpu(V_init) + beta_m_gpu(V_init));
    double h_init = alpha_h_gpu(V_init) / (alpha_h_gpu(V_init) + beta_h_gpu(V_init));
    double n_init = alpha_n_gpu(V_init) / (alpha_n_gpu(V_init) + beta_n_gpu(V_init));

    std::fill(h_m.begin(), h_m.end(), m_init);
    std::fill(h_h.begin(), h_h.end(), h_init);
    std::fill(h_n.begin(), h_n.end(), n_init);

    // Copy initial state to GPU
    CUDA_CHECK(hipMemcpy(d_data->d_V_m, h_V_m.data(), size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_data->d_m, h_m.data(), size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_data->d_h, h_h.data(), size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_data->d_n, h_n.data(), size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(d_data->d_I_inj, 0, size));
}

NeuronGPU::~NeuronGPU() {
    hipFree(d_data->d_V_m);
    hipFree(d_data->d_m);
    hipFree(d_data->d_h);
    hipFree(d_data->d_n);
    hipFree(d_data->d_I_inj);
    delete d_data;
}

void NeuronGPU::update(double dt) {
    // Copy injected currents to GPU
    size_t size = num_segments * sizeof(double);
    CUDA_CHECK(hipMemcpy(d_data->d_I_inj, injected_currents_uA.data(), size, hipMemcpyHostToDevice));

    // Configure and launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (num_segments + threadsPerBlock - 1) / threadsPerBlock;

    update_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_data->d_V_m, d_data->d_m, d_data->d_h, d_data->d_n,
        d_data->d_I_inj, num_segments, dt, d_data->g_a, d_data->surface_area
    );
    CUDA_CHECK(hipGetLastError()); // Check for kernel launch errors
    CUDA_CHECK(hipDeviceSynchronize()); // Wait for kernel to finish

    // Reset host-side injected currents
    std::fill(injected_currents_uA.begin(), injected_currents_uA.end(), 0.0);
}

void NeuronGPU::set_injected_current(int segment_index, double current_uA) {
    if (segment_index >= 0 && segment_index < num_segments) {
        injected_currents_uA[segment_index] = current_uA;
    } else {
        throw std::out_of_range("Segment index out of range.");
    }
}

double NeuronGPU::get_segment_V(int segment_index) const {
    if (segment_index < 0 || segment_index >= num_segments) {
        throw std::out_of_range("Segment index out of range.");
    }
    double V_val;
    // Copy single value from device to host
    CUDA_CHECK(hipMemcpy(&V_val, d_data->d_V_m + segment_index, sizeof(double), hipMemcpyDeviceToHost));
    return V_val;
}

std::vector<double> NeuronGPU::get_all_segment_V() const {
    std::vector<double> all_V(num_segments);
    CUDA_CHECK(hipMemcpy(all_V.data(), d_data->d_V_m, num_segments * sizeof(double), hipMemcpyDeviceToHost));
    return all_V;
}

int NeuronGPU::get_num_segments() const {
    return num_segments;
}